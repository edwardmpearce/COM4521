#include "hip/hip_runtime.h"
/* Assignment 2 Program
Building upon the code written for assignment 1, this program implements GPU code for N-body simulation and visualization.
In total, we implement a serial CPU version, an OpenMP version for multicore processors, and a CUDA version for Nvidia GPUs.
Timing code is also included for benchmarking performance.
The accompanying report provides discussion on design considerations regarding performance optimization and validation.  */
/* Problem Description
We consider a system of N bodies in frictionless 2D space exerting gravitational force on each other.
See https://en.wikipedia.org/wiki/N-body_problem for further background on the physics of the N-body problem.
We simulate the progression of such an N-body system through time using numerical integration by evaluating all pairwise
gravitational interactions between bodies in the system. The force `F_{ij}` of gravity on a body `i` exerted by a body `j`
can be calculated through the following formula: `F_{ij} = G*m_{i}*m_{j}*r_{ji}/|r_{ji}|^{3}` where `G` is the gravitational
constant, `m` denotes the mass of a body, and `r_{ji}` denotes the displacement vector from `i` towards `j`.
This is known as [Newton's Law of Universal Gravitation](https://en.wikipedia.org/wiki/Newton%27s_law_of_universal_gravitation)
We add a softening factor `eps` to the denominator to avoid the force between two approaching bodies growing without bound.
This replaces `|r_{ji}|` with `sqrt(|r_{ji}|^{2} + eps^{2})` in the expression in the denominator.
At each time `t_{k}` we calculate the resultant (sum total) force `F_{i;k}` on each body and use this to calculate 
acceleration `a_{i;k}`, then use the [Forward Euler method](https://en.wikipedia.org/wiki/Euler_method) to update the 
velocity and position at time `t_{k+1} = t_{k} + dt` based on `a_{i;k}`, `v_{i;k}`, respectively, where `dt` is the time step. */
/* C Language Library headers */
#include <stdio.h> // http://www.cplusplus.com/reference/cstdio/
#include <stdlib.h> // http://www.cplusplus.com/reference/cstdlib/
#include <string.h> // http://www.cplusplus.com/reference/cstring/
#include <ctype.h> // http://www.cplusplus.com/reference/cctype/
#include <time.h> // http://www.cplusplus.com/reference/ctime/
#include <math.h> // http://www.cplusplus.com/reference/cmath/
/* To enable OpenMP support in your project you will need to include the OpenMP header file `omp.h` 
and enable the compiler to use the OpenMP runtime. 
Set 'OpenMP Support' to 'Yes' (for both Debug and Release builds) in Project->Properties->C/C++->Language
Add `_CRT_SECURE_NO_WARNINGS` to 'Preprocessor Definitions' in Project->Properties->C/C++->Preprocessor */
#include <omp.h>
#include <hip/hip_runtime.h>
/* Local header files */
#include "NBody.h"
#include "NBodyVisualiser.h"
/* Preprocessor definitions/macros */
#define USER_NAME "smp16emp" // Replace with your username
#define BUFFER_SIZE 128 // Maximum line length accepted from input file (reasonable as only 5 (comma separated) floating point numbers expected)
#define THREADS_PER_BLOCK 256
/* Function declarations/prototypes */
void print_help();
void parseNDM(const char* argv[3]);
void parse_one_option(const char* argv[2]);
void parse_two_options(const char* argv[4]);
unsigned int parse_str_as_uint(const char* str);
void read_nbody_file(const char* filename, const int N);
void checkLastError(const char* msg);
void step_serial(void);
void step_OpenMP(void);
void step_CUDA(void);
void swap_float_pointers(float** p1, float** p2);
/* Global variables (shared by/used in multiple functions) */
/* Command line inputs */
unsigned int N; // Number of bodies in the system
unsigned int D; // Dimension of the activity grid
MODE M; // Operation mode. Allows CPU = 0, OPENMP = 1, CUDA = 2
unsigned int I = 0; // Number of iterations of the simulation to calculate when the `-i` flag is set, else 0
unsigned int f_flag = 0; // Input file flag. 0 if not specified, else such that `input_filename = options[f_flag]` in `main`.
/* Data buffers */
nbody_soa* h_nbodies; // Pointer to a structure of arrays (preferred over an array of structures for coalesced memory access)
/* Separate output buffers for updated particle positions are required to avoid interference between loop iterations/threads
when calculating forces based on current particle positions. Buffers for output velocity components are not required 
because a given particle's velocity is only used to calculate its own new position and nothing else. However this requires
each particle's new position be calculated first before its velocity is updated in-place.
Pointer swapping can be used to reduce memory copying between multiple buffers when updating system state.
See https://en.wikipedia.org/wiki/Multiple_buffering for more on double/multiple buffering 
The visualiser only (re)reads position data once after each time the simulation `step` function completes, 
rather than throughout the whole `step` calculation process, so the particles update positions in sync anyway */
/* Whether the following three pointers are host pointers or device pointers will depend on the operation mode */
float* out_x; // Pointer to store the new `x` coordinate of each body before updating in sync after loops complete
float* out_y; // Pointer to store the new `y` coordinate of each body before updating in sync after loops complete
float* activity_map; // Pointer to flattened array of D*D float values storing normalised particle density values in a 2D grid
/* Device pointers */
nbody_soa* d_nbodies; // Device pointer for nbody data

/* Device Functions and Kernels */
__device__ void swap_float_pointers(float** p1, float** p2) {
	// Function arguments are always passed by value, so to swap two pointers, we must pass references to those pointers
	// The arguments `p1` and `p2` are actually addresses of pointers to `float` data (rather than the pointers themselves)
	float* temp = *p1; // Set `temp` to be the pointer referenced by p1
	*p1 = *p2; // Overwrite the pointer addressed by `p1` with the pointer addressed by `p2`
	*p2 = temp; // Overwrite the pointer addressed by `p2` with the pointer addressed by `temp` (originally addressed by `p1`)
}

__global__ void simulation_kernel(const unsigned int N, const unsigned int D) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; // Iterating over bodies in the Nbody system, one thread per body
	if (i < N) { // One unique index for each body and any leftover threads stay idle
		float ax = 0, ay = 0; // Initialise resultant acceleration to zero
		// Read position data from global/constant/texture memory to thread-local stack variables
		float local_xi = d_nbodies->x[i];
		float local_yi = d_nbodies->y[i];
		// Calculate the acceleration of body `i` due to gravitational force from the other bodies
		for (unsigned int j = 0; j < N; j++) { 
			if (j == i) { // Skip the calculation when i = j
				continue;
			}
			// Calculate displacement from particle `i` to particle `j`, since common expression in force equation
			float x_ji = d_nbodies->x[j] - local_xi;
			float y_ji = d_nbodies->y[j] - local_yi;
			// Calculate distance from `i` to `j` with softening factor since used in denominator of force expression
			// Single precision square root: https://docs.nvidia.com/cuda/cuda-math-api/group__CUDA__MATH__SINGLE.html
			float dist_ij = sqrtf(x_ji * x_ji + y_ji * y_ji + eps_sq);
			/* Add unscaled contribution to acceleration due to gravitational force of `j` on `i`
			Universal Gravitation: `F_ij = G * m_i * m_j * r_ji / |r_ji|^3` ; Newton's 2nd Law: F_i = m_i * a_i */
			ax += d_nbodies->m[j] * x_ji / (dist_ij * dist_ij * dist_ij); // Need to scale by `G` later
			ay += d_nbodies->m[j] * y_ji / (dist_ij * dist_ij * dist_ij); // Need to scale by `G` later
			/* It would be possible to add force/acceleration contributions to `d_nbodies->v` directly within this inner loop.
			However this would cause this function to be bound by memory access latency (repeated writes to `d_nbodies->v`).
			Therefore we use the temporary/local variables `ax` and `ay` instead */
		}
		/* Use current velocity, acceleration to calculate position, velocity at next time step, respectively. */
		float local_vxi = d_nbodies->vx[i];
		float local_vyi = d_nbodies->vy[i];
		// Care has to be taken about the order of execution to ensure the output positions are calculated correctly
		// Use current velocity to calculate next position
		local_xi += local_vxi * dt;
		local_yi += local_vyi * dt;
		// Now the local position variables hold the new positions and can be used to update the activity map

		// Use current acceleration (based on current positions) to calculate the new velocity
		// Scale `ax`, `ay` by gravitational constant `G`. See `NBody.h` for definition and comment.
		d_nbodies->vx[i] = local_vxi + G * ax * dt; // Write the new velocity back to `d_nbodies->vx[i]`
		d_nbodies->vy[i] = local_vyi + G * ay * dt; // Write the new velocity back to `d_nbodies->vy[i]`
		// We can update particle velocities in-place without adversely affecting subsequent iterations/other threads

		// Write the new position of particle `i` to the output buffers to avoid interfering with other threads
		out_x[i] = local_xi;
		out_y[i] = local_yi;
		// Pointer swapping of position data occurs outside of the kernel launch in the `step_CUDA` function

		// Update the activity map - a flat array of D*D float values storing normalised particle density values in a 2D grid
		// First check whether the new position of particle `i` is within the activity grid [0,1)^{2}. Branching thread logic.
		if ((local_xi >= 0) && (local_xi < 1) && (local_yi >= 0) && (local_yi < 1)) {
			// If so, calculate the index of the grid element that particle `i` is in
			// Multiply position vector by `D` then truncate components to `int` to find position in \{0,...,D-1\}^{2} grid
			unsigned int index = D * (int)(D * local_yi) + (int)(D * local_xi); // Linearize the index from 2D grid into 1D array

			// Increase the associated histogram bin by the normalised quantity `D/N` (scaling by D to increase brightness)
			// Can result in race condition as multiple threads could increment at once. Could solve with `atomicAdd`
			activity_map[index] += (float) D / N; 
			// Unfortunately this is a random access (write) to global memory and cannot easily be coalesced
			/* We choose not to reduce the number of multiplication/division operations by incrementing the histogram bin by one
			at this step and then scaling the histogram counts in a separate loop (as in the other implementations) in order to
			avoid launching a separate grid/kernel with D^2 threads, thus reducing the number of kernel launchs */
		}
	}
}

/* For information on how to parse command line parameters, see http://www.cplusplus.com/articles/DEN36Up4/ 
`argc` in the count of the command arguments, and `argv` is an array (of length `argc`) of the arguments. 
The first argument is always the executable name (including path) */
int main(const int argc, const char *argv[]) {
	/* Process the command line arguments */
	switch (argc) {
	case 4: // No optional flags used
		parseNDM(&argv[1]);
		break;
	case 6: // One optional flag and argument used
		parse_one_option(&argv[4]);
		parseNDM(&argv[1]);
		break;
	case 8: // Two optional flags with arguments used
		parse_two_options(&argv[4]);
		parseNDM(&argv[1]);
		break;
	default: // The expected arguments are: "nbody.exe N D M [-i I] [-f input_file]"
		fprintf(stderr, "Error: Unexpected number of arguments. %d arguments (including executable name) received\n", argc);
		print_help();
		exit(EXIT_FAILURE);
	}

	// Declare a function pointer to a simulation step function and set its value according to the operation mode `M`
	void (*simulate)(void) = NULL; // Declare `simulate` as pointer to function (void) returning void
	switch (M) {
	case CPU:
		simulate = &step_serial;
		break;
	case OPENMP:
		simulate = &step_OpenMP;
		printf("OpenMP using %d threads\n", omp_get_max_threads());
		break;
	case CUDA:
		simulate = &step_CUDA;
		break;
	}

	/* Allocate Heap Memory */
	// Calculate memory requirements
	const unsigned int data_column_size = sizeof(float) * N;
	const unsigned int activity_grid_size = sizeof(float) * D * D;

	// Memory allocation. See http://www.cplusplus.com/reference/cstdlib/malloc/
	h_nbodies = (nbody_soa*)malloc(sizeof(nbody_soa));
	h_nbodies->x = (float*)malloc(data_column_size);
	h_nbodies->y = (float*)malloc(data_column_size);
    // Allocates memory block for length N array of floats, and initialize all bits to zero (for default zero initial velocity).
	// See http://www.cplusplus.com/reference/cstdlib/calloc/
	h_nbodies->vx = (float*)calloc(N, sizeof(float)); // Zero initial velocity
	h_nbodies->vy = (float*)calloc(N, sizeof(float)); // Zero initial velocity
	h_nbodies->m = (float*)malloc(data_column_size);
	if ((h_nbodies == NULL) || (h_nbodies->x == NULL) || (h_nbodies->y == NULL) || (h_nbodies->vx == NULL) || (h_nbodies->vy == NULL) || (h_nbodies->m == NULL)) {
		fprintf(stderr, "Error allocating host memory (`h_nbodies`) for system with %d bodies\n", N);
		exit(EXIT_FAILURE);
	}
	if (M == CUDA) {
		/* Allocate device memory */
		hipMalloc((void**)&d_nbodies, sizeof(nbody_soa));
		hipMalloc((void**)&d_nbodies->x, data_column_size);
		hipMalloc((void**)&d_nbodies->y, data_column_size);
		hipMalloc((void**)&d_nbodies->vx, data_column_size);
		hipMalloc((void**)&d_nbodies->vy, data_column_size);
		hipMalloc((void**)&d_nbodies->m, data_column_size);
		hipMalloc((void**)&out_x, data_column_size);
		hipMalloc((void**)&out_y, data_column_size);
		hipMalloc((void**)&activity_map, activity_grid_size);
		checkCUDAError("Memory allocation on device with hipMalloc");
	}
	else { // Whether `out_x`, `out_y`, and `activity_map` are pointers on the host or device depends on operation mode
		out_x = (float*)malloc(data_column_size);
		out_y = (float*)malloc(data_column_size);
		if ((out_x == NULL) || (out_y == NULL)) {
			fprintf(stderr, "Error allocating host memory (output position buffers) for system with %d bodies\n", N);
			exit(EXIT_FAILURE);
		}
		activity_map = (float*)malloc(activity_grid_size);
		if (activity_map == NULL) {
			fprintf(stderr, "Error allocating host memory (`activity map`) for system with %d bodies, activity grid size %d\n", N, D);
			exit(EXIT_FAILURE);
		}
	}

	/* Read initial data from file to host memory, or generate random initial state according to optional program flag `-f`. */
	if (f_flag == 0) { // No input file specified, so a random initial N-body state will be generated
		const float one_over_N = (float)1 / N; // Store the inverse of `N` as a constant to avoid recalculating in loop
		for (unsigned int i = 0; i < N; i++) {
			h_nbodies->x[i] = (float)rand() / RAND_MAX; // Random position in [0,1]
			h_nbodies->y[i] = (float)rand() / RAND_MAX; // Random position in [0,1]
			h_nbodies->m[i] = one_over_N; // Mass distributed equally among N bodies
			// Note that velocity data has already been initialized to zero for all bodies
		}
	}
	else { // Attempt to read initial N-body system state from input csv file to host memory
		read_nbody_file(argv[f_flag], N);
	}

	if (M == CUDA) {
	/* Copy the host input values in `h_nbodies` to the device memory `d_nbodies`. */
		hipMemcpy(d_nbodies->x, h_nbodies->x, data_column_size, hipMemcpyHostToDevice);
		hipMemcpy(d_nbodies->y, h_nbodies->y, data_column_size, hipMemcpyHostToDevice);
		hipMemcpy(d_nbodies->vx, h_nbodies->vx, data_column_size, hipMemcpyHostToDevice);
		hipMemcpy(d_nbodies->vy, h_nbodies->vy, data_column_size, hipMemcpyHostToDevice);
		hipMemcpy(d_nbodies->m, h_nbodies->m, data_column_size, hipMemcpyHostToDevice);
		checkCUDAError("Input transfer to device");
	}

	/* According to the value of program argument `I` either configure and start the visualiser, 
	or perform a fixed number of simulation steps and output the timing results. */
	if (I == 0) { // Run visualiser when number of iterations not specified with `-i` flag, or otherwise `I` was set to 0
		initViewer(N, D, M, simulate); // The simulation step function has been set earlier according to operation mode `M`
		// Set where the visualiser will check for particle position data after each iteration
		if (M == CUDA) {
			setNBodyPositions(d_nbodies); // Device pointer
		}
		else {
			setNBodyPositions(h_nbodies); // Host pointer
		}
		setActivityMapData(activity_map); // This is where the visualiser will check for activity data after each iteration
		startVisualisationLoop();
	}
	else { // Run the simulation for `I` iterations and output the timing results
		switch (M) { // Simulation and timing methods depend on operation mode
		case CPU:
			clock_t t; // Clock ticks for serial CPU timing
			double seconds = 0; // Variable to hold execution timing results
			t = clock(); // Starting timestamp. See http://www.cplusplus.com/reference/ctime/clock/
			for (unsigned int i = 0; i < I; i++) {
				step_serial();
			}
			t = clock() - t; // Take end timestamp and calculate difference from start in clock ticks
			seconds = (double)t / CLOCKS_PER_SEC;
			printf("Execution time %d seconds %d milliseconds for %d iterations\n", (int)seconds, (int)((seconds - (int)seconds) * 1000), I);
			break;
		case OPENMP:
			double start, end; // Timestamps for OpenMP timing
			double seconds = 0; // Variable to hold execution timing results
			start = omp_get_wtime(); // Starting timestamp. See https://www.openmp.org/spec-html/5.0/openmpsu160.html
			for (unsigned int i = 0; i < I; i++) {
				step_OpenMP();
			}
			end = omp_get_wtime();
			seconds = end - start;
			printf("Execution time %d seconds %d milliseconds for %d iterations\n", (int)seconds, (int)((seconds - (int)seconds) * 1000), I);
			break;
		case CUDA:
			hipEvent_t cuda_start, cuda_stop; // CUDA Event timers
			float milliseconds = 0; // Timing results variable (must be `float` type for call to `hipEventElapsedTime`)
			hipEventCreate(&cuda_start); hipEventCreate(&cuda_stop); // Create CUDA Events
			hipEventRecord(cuda_start); // Record the start time before calling the kernel launching simulation function
			for (unsigned int i = 0; i < I; i++) {
				step_CUDA();
			}
			hipEventRecord(cuda_stop); // Record the stop time once the simulation has finished
			hipEventSynchronize(cuda_stop); // Ensure stop time has finished recording before continuing
			checkCUDAError("Error running simulation kernel\n");
			hipEventElapsedTime(&milliseconds, cuda_start, cuda_stop);	// Write the elapsed time to `milliseconds`
			printf("Execution time %d seconds %d milliseconds for %d iterations\n", (int)milliseconds / 1000, (int)milliseconds % 1000, I);
			hipEventDestroy(cuda_start); hipEventDestroy(cuda_stop); // Cleanup CUDA Event timers
			/* Copy the device output values in `d_nbodies` to the host memory `h_nbodies` then write to file for validation.
			hipMemcpy(h_nbodies->x, d_nbodies->x, data_column_size, hipMemcpyDeviceToHost);
			hipMemcpy(h_nbodies->y, d_nbodies->y, data_column_size, hipMemcpyDeviceToHost);
			hipMemcpy(h_nbodies->vx, d_nbodies->vx, data_column_size, hipMemcpyDeviceToHost);
			hipMemcpy(h_nbodies->vy, d_nbodies->vy, data_column_size, hipMemcpyDeviceToHost);
			checkCUDAError("Copying final Nbody system state from device to host"); 
			*/
			break;
		}
	}

	// Cleanup
	if (M == CUDA) {
		hipFree(d_nbodies->x);
		hipFree(d_nbodies->y);
		hipFree(d_nbodies->vx);
		hipFree(d_nbodies->vy);
		hipFree(d_nbodies->m);
		hipFree(d_nbodies);
		hipFree(out_x);
		hipFree(out_y);
		hipFree(activity_map);
		checkCUDAError("Freeing memory from device with hipFree");
	}
	else { // Whether `out_x`, `out_y`, and `activity_map` are pointers on the host or device depends on operation mode
		free(out_x);
		free(out_y);
		free(activity_map);
	}
	free(h_nbodies->x);
	free(h_nbodies->y);
	free(h_nbodies->vx);
	free(h_nbodies->vy);
	free(h_nbodies->m);
	free(h_nbodies);

	return 0;
}

/* Functions to perform the main simulation of the Nbody system by updating the state by one time step */
// Serial CPU version
void step_serial(void) {
	/* The index `i` is used to iterate over the `N` bodies in the system. For each body `i`, we choose to calculate the
	`N-1` interactions of the other bodies `j != i` on `i`, as opposed to the action of `i` on all of the other bodies `j != i`.
	When computed in parallel, the former avoids a synchronisation step so that the velocity of each body `i`
	can be updated independently of the other threads, reducing idle time. Afterwards, we can also update 
	the position of body `i` and calculate which histogram bin/activity grid cell the body `i` is in, all within one loop.
	This is known as loop jamming or loop fusion. See http://www.it.uom.gr/teaching/c_optimization/tutorial.html
	Calculating the histogram contribution of each body is far more efficient than iterating over histogram bins/grid cells
	since we exploit the fact that each body can only be in at most one grid cell at a time (D*D times fewer calculations). */
	unsigned int i, j; // Counter variables
	float ax, ay; // Components of resultant acceleration of a particle as a result of gravitational force
	float local_xi, local_yi; // Local position variables to reduce global memory accesses, especially during inner loop
	float local_vxi, local_vyi; // Local velocity variables to exchange two global memory reads for one plus two local reads
	float x_ji, y_ji; // Components of displacement vector from particle `i` to particle `j`
	float dist_ij; // To hold softened distance `sqrt(|r_{ji}|^{2} + eps^{2})` from `i` to `j`

	// Reset histogram values to zero with `memset`. See http://www.cplusplus.com/reference/cstring/memset/
	memset(activity_map, 0, sizeof(activity_map));

	for (i = 0; i < N; i++) { // Iterating over bodies in the Nbody system
		ax = 0; // Reset resultant acceleration in `x` direction to zero for new particle
		ay = 0; // Reset resultant acceleration in `y` direction to zero for new particle
		// Read position data from global memory to the stack
		local_xi = h_nbodies->x[i];
		local_yi = h_nbodies->y[i];

		for (j = 0; j < N; j++) {
			if (j == i) { // Skip the calculation when i = j (saves calculation time; could consider branching effects on GPU)
				continue;
			}
			// Calculate displacement from particle `i` to particle `j`, since common expression in force equation
			// Using local variables for `x[i]`, `y[i]` here removes a global memory read from each inner loop iteration
			x_ji = h_nbodies->x[j] - local_xi;
			y_ji = h_nbodies->y[j] - local_yi;
			// Calculate distance from `i` to `j` with softening factor since used in denominator of force expression
			// Explicit casting required since `sqrt` function expects `double` type input and output; operation execution order
			dist_ij = (float)sqrt((double)x_ji * x_ji + (double)y_ji * y_ji + eps_sq);
			/* Add unscaled contribution to acceleration due to gravitational force of `j` on `i`
			Universal Gravitation: `F_ij = G * m_i * m_j * r_ji / |r_ji|^3` ; Newton's 2nd Law: F_i = m_i * a_i
			See top of file for further explanation of calculation, physical background */
			ax += h_nbodies->m[j] * x_ji / (dist_ij * dist_ij * dist_ij); // Need to scale by `G` later
			ay += h_nbodies->m[j] * y_ji / (dist_ij * dist_ij * dist_ij); // Need to scale by `G` later
			/* It would be possible to add force/acceleration contributions to `h_nbodies->v` directly within this inner loop.
			However this would cause this function to be bound by memory access latency (repeated writes to `h_nbodies->v`).
			Therefore we use the temporary/local variables `ax` and `ay` instead */
		}
		/* Use current velocity, acceleration to calculate position, velocity at next time step, respectively. */
		/* Former code uses extra heap memory buffers for velocity, adding extra steps pointer swapping and using more memory
		However this implementation scores highly for readability, as it makes the intended outcome clear (no race conditions)
		out_x[i] = h_nbodies->x[i] + h_nbodies->vx[i] * dt;
		out_y[i] = h_nbodies->y[i] + h_nbodies->vy[i] * dt;
		out_vx[i] = h_nbodies->vx[i] + G * ax * dt;
		out_vy[i] = h_nbodies->vy[i] + G * ay * dt; */
		// Using local velocity variables also reduces global memory reads, but only marginally compared `local_xi`, `local_yi`
		local_vxi = h_nbodies->vx[i];
		local_vyi = h_nbodies->vy[i];
		// More care has to be taken about the order of execution to ensure the output positions are calculated correctly
		// Use current velocity to calculate next position
		local_xi += local_vxi * dt;
		local_yi += local_vyi * dt;
		// Now the local position variables hold the new positions and can be used to update the activity map
		// Use current acceleration (based on current positions) to calculate the new velocity
		// Scale `ax`, `ay` by gravitational constant `G`. See `NBody.h` for definition and comment.
		h_nbodies->vx[i] = local_vxi + G * ax * dt; // Write the new velocity back to `h_nbodies->vx[i]`
		h_nbodies->vy[i] = local_vyi + G * ay * dt; // Write the new velocity back to `h_nbodies->vy[i]`
		// We can update particle velocities in-place without adversely affecting subsequent iterations/other threads

		// Update the activity map - a flat array of D*D float values storing normalised particle density values in a 2D grid
		// First check whether the new position of particle `i` is within the activity grid [0,1)^{2}
//		if ((out_x[i] >= 0) && (out_x[i] < 1) && (out_y[i] >= 0) && (out_y[i] < 1)) {
		if ((local_xi >= 0) && (local_xi < 1) && (local_yi >= 0) && (local_yi < 1)) {
			// If so, calculate the index of the grid element that particle `i` is in and increment the associated histogram bin
			// Multiply position vector by `D` then truncate components to `int` to find position in \{0,...,D-1\}^{2} grid
			// Can result in race condition when outer `i` loop parallel as multiple threads could increment at once
			// Possible solutions: Critical section; atomic operator; move section outside parallel loop (barrier/master method)
			activity_map[D * (int)(D * local_yi) + (int)(D * local_xi)]++; // Linearize the index from 2D grid into 1D array
		}
		// Write the new position of particle `i` to the output buffers to avoid interfering with subsequent iterations
		out_x[i] = local_xi;
		out_y[i] = local_yi;
	}
	// Scale activity map values by `D / N` to normalize the histogram values and then scale by D to increase brightness
	const float one_over_N = (float)1 / N; // Store the inverse of global variable `N` as a constant to cache value
	for (i = 0; i < D * D; i++) {
		activity_map[i] *= one_over_N * D;
	}
	/* Finally, update the `nbody` data pointers to reference the newly calculated arrays of position data.
	We swap the input and output pointers rather than simply overwriting the input pointers because that would result
	in losing the original input pointers, losing allocated heap memory addresses and causing a memory leak! */
	float* temp; // Declare a temporary pointer to `float` to hold addresses whilst swapping the input and output pointers
	temp = h_nbodies->x; // Keep track of the old input pointer for later use so we don't lose any allocated memory
	h_nbodies->x = out_x; // Update the `h_nbodies->x` pointer which is used for visualisation, and the next `step` iteration
	out_x = temp; // Reset `out_x` to a 'fresh', 'empty' piece of memory
	temp = h_nbodies->y; // Keep track of the old input pointer for later use so we don't lose any allocated memory
	h_nbodies->y = out_y; // Update the `h_nbodies->y` pointer which is used for visualisation, and the next `step` iteration
	out_y = temp; // Reset `out_y` to a distinct piece of 'fresh' and 'empty' memory
}

/* Profiling with Visual Studio's Diagnostic Tools and PerfTips by setting breakpoints to time code segments and using
Debug->Windows->Show Diagnostic Tools https://docs.microsoft.com/en-us/visualstudio/profiling/profiling-feature-tour?view=vs-2019
Shows that as `N` increases the majority of time spent running the programme is spent calling the simulation step function,
and within that the loop over `N` particles (indexed by `i`) occupies most of the compute time rather than the loop over
the activity grid cells. This makes sense since there are far more compute steps within the `i` loop, and generally D will be
much smaller than `N` and is effectively limited in visualisation by screen resolution.
Therefore it is most important to parallelise the outer `i` loop. Further analysis suggests that as `N` increases further,
the majority of time spent within each outer loop is spent iterating over the inner `j` loop of interactions between particles,
so nested parallel loops should also be considered. Amongst all the operations/function calls within each simulation step,
it appears that the call to the `sqrt` function in the inner loop is the most expensive. */

// OpenMP version (For parallel computation on a multicore CPU)
/* Benchmarking results for parallelising outer loop over `i` (on my 4 core personal laptop)
Command Line Arguments | Histogram Race Handling | Scheduling | Execution Time(s)
"16384 16 CPU -i 10" | N/A | N/A | 61.357s, 61.988s
"16384 16 OPENMP -i 10" | Atomic | schedule(static) | 17.184s, 17.624s, 17.713s
"16384 16 OPENMP -i 10" | Atomic | schedule(static, 1) | 17.314s, 17.399s
"16384 16 OPENMP -i 10" | Atomic | schedule(static, 2) | 17.182s, 17.478s
"16384 16 OPENMP -i 10" | Atomic | schedule(static, 4) | 17.225s, 17.363s
"16384 16 OPENMP -i 10" | Atomic | schedule(static, 8) | 17.248s, 17.446s
"16384 16 OPENMP -i 10" | Atomic | schedule(guided) | 15.769s, 16.650s, 16.830s, 17.110s
"16384 16 OPENMP -i 10" | Atomic | schedule(dynamic) | 11.895s, 12.028s, 12.210s
"16384 16 OPENMP -i 10" | Atomic | schedule(dynamic, 2) | 11.841s, 11.997s, 12.266s
"16384 16 OPENMP -i 10" | Atomic | schedule(dynamic, 4) | 11.766s, 12.047s, 12.210s
"16384 16 OPENMP -i 10" | Critical | schedule(static, 4) | 17.341s, 18.007s
"16384 16 OPENMP -i 10" | Critical | schedule(guided) | 16.720s, 16.994s
"16384 16 OPENMP -i 10" | Critical | schedule(dynamic) | 11.617s, 11.888s, 11.999s, 13.054s
"8192 16 CPU -i 100" | N/A | N/A | 153.307s, 154.100s
"8192 16 OPENMP -i 100" | Atomic | schedule(dynamic) | 29.687s, 30.332s
"8192 16 OPENMP -i 100" | Critical | schedule(dynamic) | 28.490s, 30.305s
"2048 16 CPU -i 1000" | N/A | N/A | 95.132s, 95.174s, 95.677s
"2048 16 OPENMP -i 1000" | Atomic | schedule(static) | 29.240s, 29.584s
"2048 16 OPENMP -i 1000" | Atomic | schedule(static, 4) | 29.685s, 30.073s
"2048 16 OPENMP -i 1000" | Atomic | schedule(guided) | 29.357s, 29.521s
"2048 16 OPENMP -i 1000" | Atomic | schedule(dynamic) | 17.910s, 18.774s
"2048 16 OPENMP -i 1000" | Critical | schedule(dynamic) | 18.470s, 18.577s
"512 16 CPU -i 10000" | N/A | N/A | 59.003s, 59.404s
"512 16 OPENMP -i 10000" | Atomic | schedule(dynamic) | 12.734s, 13.735s
"256 16 CPU -i 100000" | N/A | N/A | 148.552s, 149.194s
"256 16 OPENMP -i 100000" | Atomic | schedule(static, 4) | 47.987s, 48.889s
"256 16 OPENMP -i 100000" | Atomic | schedule(dynamic) | 34.568s, 36.429s
"128 16 CPU -i 100000" | N/A | N/A | 37.436s, 37.520s
"128 16 OPENMP -i 100000" | Atomic | schedule(static, 4) | 12.587s, 13.110s
"128 16 OPENMP -i 100000" | Atomic | schedule(guided) | 12.448s, 12.541s
"128 16 OPENMP -i 100000" | Atomic | schedule(dynamic) | 11.717s, 12.071s
"64 16 CPU -i 1000000" | N/A | N/A | 92.477s, 92.617s
"64 16 OPENMP -i 1000000" | Atomic | schedule(static, 4) | 31.302s, 32.235s
"64 16 OPENMP -i 1000000" | Atomic | schedule(guided) | 32.292s, 33.169s
"64 16 OPENMP -i 1000000" | Atomic | schedule(dynamic) | 38.062s, 38.133s */
/* Initial Remarks on parallelising outer loop
The data shows that dynamic scheduling is faster for values of `N` greater than 100 or so, but slower than static scheduling for
smaller values on `N`, with guided scheduling always performing between to static and dynamic scheduling and never optimal. 
This is because there is uneven workload amongst threads, which favours dynamic scheduling, but the overhead cost of dynamic 
scheduling at runtime becomes a limiting factor for relatively small parallel loops.
A trend of increasing OpenMP performance relative to serial CPU performance as `N` increases can also be seen as the benefits
of parallelism outweigh their overhead costs.
I believe the main source of difference in workload between threads arises from whether the particle `i` lies within 
the activity grid or not. If so, a slow (atomic/critical/serial) incrementation of an activity grid cell must occur, which also
involves writing to global memory at an index of the `activity_map` array that cannot be predicted at compile time, but if
the particle `i` lies outside the activity grid this step can be skipped, causing an uneven workload between different threads.
Scheduling approach seems to have a more important impact on performance than chunk size.
On the other hand, it appears that there's no major difference in performance between using a critical section or an atomic
directive to ensure the safe incrementation of the activity grid histogram, perhaps with only a slight leaning towards atomic.
For reference information on the OMP Atomic directive, see the following links:
https://www.openmp.org/spec-html/5.0/openmpsu95.html
https://www.ibm.com/support/knowledgecenter/SSGH2K_13.1.2/com.ibm.xlc131.aix.doc/compiler_ref/prag_omp_atomic.html */
/* Benchmarking results for parallelising inner loop over `j` only (on my 4 core personal laptop)
Command Line Arguments | Acceleration Sum Handling | Scheduling | Execution Time(s)
"8192 16 OPENMP -i 10" | Two Atomic Directives | schedule(static) | 54.309s
"8192 16 OPENMP -i 10" | Two Atomic Directives | schedule(dynamic) | 49.997s
"8192 16 OPENMP -i 10" | Critical Section | schedule(static) | 54.520s
"8192 16 OPENMP -i 10" | Critical Section | schedule(dynamic) | 61.185s
"256 16 OPENMP -i 10000" | Two Atomic Directives | schedule(static) | 57.011s
"256 16 OPENMP -i 10000" | Two Atomic Directives | schedule(static, 4) | 64.039s
"256 16 OPENMP -i 10000" | Two Atomic Directives | schedule(dynamic) | 90.021s
"256 16 OPENMP -i 10000" | Critical Section | schedule(static) | 58.980s
"256 16 OPENMP -i 10000" | Critical Section | schedule(dynamic) | 79.218s */
/* Remarks on parallelising inner loop
The data shows that only parallelising the inner `j` loop over force interactions results in a 3-4x slowdown 
compared to the serial CPU version. This is because of repeated overheads setting up small parallel loops within a larger loop.
The story might be different on a machine with more cores (e.g. 16 cores rather than 4 cores), but when compared to the 
3-4x speedup over serial implementation from parallelising the outer `i` loop over bodies in the system, it is clear which is
preferred. As a general rule, outer loops should be parallelised first (assuming they run for a reasonable number of iterations).
Final remarks and conclusion
Finally, through testing an implementation of nested parallel loops we find the following performance heirarchy for the given
problem: Parallel outer loop > Serial CPU version > Parallel histogram scaling > Nested parallel loops > Parallel inner loop.
For **nested** parallel loops with dynamic scheduling and atomic directives to avoid race conditions when 
1) Incrementing activity map contributions; and 2) Summing acceleration contributions with the inner `j` loop parallel;
given command line arguments "8192 16 OPENMP -i 100", an execution time of 268.809s was recorded, about 75% slower than serial. 
In conclusion, we choose to parallelise the force calculation outer loop over `i` iterating over bodies in the system as it
is the best and only loop parallelisation which improves on the serial CPU version (by a respectable 3-6x speedup), 
we choose dynamic scheduling since it outperforms static and guided scheduling for values of `N` greater than around 100, where
many feasible values of `N` lie (a separate parallel directive to choose static scheduling when N < 100 could be considered).
Finally, to avoid race conditions when each thread uses the position of its local particle to update the shared `activity_map`
histogram, we choose to use an atomic directive, though this only appears to be negligably better than a critical section. */
void step_OpenMP(void) {
	/* The index `i` is used to iterate over the `N` bodies in the system. For each body `i`, we choose to calculate the
	`N-1` interactions of the other bodies `j != i` on `i`, as opposed to the action of `i` on all of the other bodies `j != i`.
	This is because the latter requires an extra synchronisation step before the velocity of each body `i` can be calculated,
	increasing thread idle time. Subsequently, we can also update the position of body `i` and calculate its activity grid
	position within the same parallel loop, reducing overhead. This is known as loop jamming or loop fusion. 
	See http://www.it.uom.gr/teaching/c_optimization/tutorial.html
	Calculating the histogram contribution of each body is far more efficient than iterating over histogram bins/grid cells
	since we exploit the fact that each body can only be in at most one grid cell at a time (D*D times fewer calculations). */
	int i, j; // Counter variables. OpenMP requires these to be `int` type rather than unsigned
	float ax, ay; // Components of resultant acceleration of a particle as a result of gravitational force
	float local_xi, local_yi; // Local position variables to reduce global memory accesses, especially during inner loop
	float local_vxi, local_vyi; // Local velocity variables to exchange two global memory reads for one plus two local reads
	float x_ji, y_ji; // Components of displacement vector from particle `i` to particle `j`
	float dist_ij; // To hold softened distance `sqrt(|r_{ji}|^{2} + eps^{2})` from `i` to `j`

	// Reset histogram values to zero with `memset`. See http://www.cplusplus.com/reference/cstring/memset/
	memset(activity_map, 0, sizeof(activity_map));

	//omp_set_nested(1);
#pragma omp parallel for default(none) private(i, j, ax, ay, local_xi, local_yi, x_ji, y_ji, dist_ij, local_vxi, local_vyi) shared(h_nbodies, activity_map, D, out_x, out_y) schedule(dynamic)
	for (i = 0; i < N; i++) { // Iterating over bodies in the Nbody system
		ax = 0; // Reset resultant acceleration in `x` direction to zero for new particle
		ay = 0; // Reset resultant acceleration in `y` direction to zero for new particle
		// Read position data from global memory to the stack
		local_xi = h_nbodies->x[i];
		local_yi = h_nbodies->y[i];

// Can treat `i` as a shared variable on the inner `j` loop since we read without changing within each outer loop iteration
// Otherwise could use `firstprivate(i)` declaration to pass in the value to each thread
//#pragma omp parallel for default(none) private(j, x_ji, y_ji, dist_ij) shared(i, ax, ay, local_xi, local_yi, h_nbodies) schedule(dynamic)
		for (j = 0; j < N; j++) {
			if (j == i) { // Skip the calculation when i = j (saves calculation time; could consider branching effects on GPU)
				continue;
			}
			// Calculate displacement from particle `i` to particle `j`, since common expression in force equation
			// Using local variables for `x[i]`, `y[i]` here removes a global memory read from each inner loop iteration
			x_ji = h_nbodies->x[j] - local_xi;
			y_ji = h_nbodies->y[j] - local_yi;
			// Calculate distance from `i` to `j` with softening factor since used in denominator of force expression
			// Explicit casting required since `sqrt` function expects `double` type input and output; operation execution order
			dist_ij = (float)sqrt((double)x_ji * x_ji + (double)y_ji * y_ji + eps_sq);
			/* Add unscaled contribution to acceleration due to gravitational force of `j` on `i`
			Universal Gravitation: `F_ij = G * m_i * m_j * r_ji / |r_ji|^3` ; Newton's 2nd Law: F_i = m_i * a_i
			See top of file for further explanation of calculation, physical background */
			// If the inner `j` loop is parallel, adding to `ax[i]` will result in a race condition. 
			// Could try a reduction directive for `ax`, `ay` in the parallel inner loop directive if supported by OpenMP 2.0
			//#pragma omp critical {
			ax += h_nbodies->m[j] * x_ji / (dist_ij * dist_ij * dist_ij); // Need to scale by `G` later
			ay += h_nbodies->m[j] * y_ji / (dist_ij * dist_ij * dist_ij); // Need to scale by `G` later
			//}
			/* It would be possible to add force/acceleration contributions to `h_nbodies->v` directly within this inner loop.
			However this would cause this function to be bound by memory access latency (repeated writes to `h_nbodies->v`).
			Therefore we use the temporary/local variables `ax` and `ay` instead */
		}
		/* Use current velocity, acceleration to calculate position, velocity at next time step, respectively. */
		/* Former code uses extra heap memory buffers for velocity, adding extra steps pointer swapping and using more memory
		However this implementation scores highly for readability, as it makes the intended outcome clear (no race conditions)
		out_x[i] = h_nbodies->x[i] + h_nbodies->vx[i] * dt;
		out_y[i] = h_nbodies->y[i] + h_nbodies->vy[i] * dt;
		out_vx[i] = h_nbodies->vx[i] + G * ax * dt;
		out_vy[i] = h_nbodies->vy[i] + G * ay * dt; */
		// Using local velocity variables also reduces global memory reads, but only marginally compared `local_xi`, `local_yi`
		local_vxi = h_nbodies->vx[i];
		local_vyi = h_nbodies->vy[i];
		// More care has to be taken about the order of execution to ensure the output positions are calculated correctly
		// Use current velocity to calculate next position
		local_xi += local_vxi * dt;
		local_yi += local_vyi * dt;
		// Now the local position variables hold the new positions and can be used to update the activity map
		// Use current acceleration (based on current positions) to calculate the new velocity
		// Scale `ax`, `ay` by gravitational constant `G`. See `NBody.h` for definition and comment.
		h_nbodies->vx[i] = local_vxi + G * ax * dt; // Write the new velocity back to `h_nbodies->vx[i]`
		h_nbodies->vy[i] = local_vyi + G * ay * dt; // Write the new velocity back to `h_nbodies->vy[i]`
		// We can update particle velocities in-place without adversely affecting subsequent iterations/other threads

		// Update the activity map - a flat array of D*D float values storing normalised particle density values in a 2D grid
		// First check whether the new position of particle `i` is within the activity grid [0,1)^{2}
//		if ((out_x[i] >= 0) && (out_x[i] < 1) && (out_y[i] >= 0) && (out_y[i] < 1)) {
		if ((local_xi >= 0) && (local_xi < 1) && (local_yi >= 0) && (local_yi < 1)) {
			// If so, calculate the index of the grid element that particle `i` is in and increment the associated histogram bin
			// Multiply position vector by `D` then truncate components to `int` to find position in \{0,...,D-1\}^{2} grid
			// Can result in race condition when outer `i` loop parallel as multiple threads could increment at once
			// Possible solutions: Critical section; atomic operator; move section outside parallel loop (barrier/master method)
			/* Atomic operations can be used to safely increment a shared numeric value; critical regions have other uses too */
			#pragma omp atomic
			activity_map[D * (int)(D * local_yi) + (int)(D * local_xi)]++; // Linearize the index from 2D grid into 1D array
		}
		// Write the new position of particle `i` to the output buffers to avoid interfering with other threads/iterations
		out_x[i] = local_xi;
		out_y[i] = local_yi;
	}
	// Scale activity map values by `D / N` to normalize the histogram values and then scale by D to increase brightness
	const float one_over_N = (float)1 / N; // Store the inverse of global variable `N` as a constant to cache value
	/* Parallelising this histogram scaling loop actually has a negative impact on performance due to fork/join overheads
	outweighing the small gains from parallelising a non-compute intensive loop. Using command line arguments (release mode)
	"2048 1024 OPENMP -i 100" we reliably time 9.8 seconds for serial execution vs 11.5s-11.9s with this loop parallel 
	and using static or guided scheduling (chunk size has little effect) and 14.5s-14.9s for dynamic scheduling. 
	The reason dynamic scheduling is even slower than static scheduling is the extra runtime overheads of dynamic scheduling
	where the workloads are extremely uniform (two multiplications per loop) */
//#pragma omp parallel for default(none) private(i) shared(activity_map, one_over_N, D) schedule(dynamic)
	for (i = 0; i < D * D; i++) {
		activity_map[i] *= one_over_N * D;
	}
	/* Finally, update the `nbody` data pointers to reference the newly calculated arrays of position data.
	We swap the input and output pointers rather than simply overwriting the input pointers because that would result
	in losing the original input pointers, losing allocated heap memory addresses and causing a memory leak! */
	float* temp; // Declare a temporary pointer to `float` to hold addresses whilst swapping the input and output pointers
	temp = h_nbodies->x; // Keep track of the old input pointer for later use so we don't lose any allocated memory
	h_nbodies->x = out_x; // Update the `h_nbodies->x` pointer which is used for visualisation, and the next `step` iteration
	out_x = temp; // Reset `out_x` to a 'fresh', 'empty' piece of memory
	temp = h_nbodies->y; // Keep track of the old input pointer for later use so we don't lose any allocated memory
	h_nbodies->y = out_y; // Update the `h_nbodies->y` pointer which is used for visualisation, and the next `step` iteration
	out_y = temp; // Reset `out_y` to a distinct piece of 'fresh' and 'empty' memory
}

// CUDA version (for parallel computation on GPU)
void step_CUDA(void) {
	/* This host function sets up kernel launch parameters and launches GPU kernel(s) to calculate one simulation step */
	// First reset histogram values to zero with `hipMemset`. 
	// See https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__MEMORY.html for documentation
	hipMemset(activity_map, 0, sizeof(activity_map));

	// Prepare kernel launch parameters
	unsigned int blocks = N / THREADS_PER_BLOCK;
	if ((N % THREADS_PER_BLOCK) != 0) { // Ensure we have the minimum number of blocks needed for total threads to exceed `N`
		blocks++;
	}
	// Run the kernel
	simulation_kernel << <blocks, 256 >> > (N, D);
	//checkCUDAError("Error running simulation kernel\n");

	/* New velocities and activity map data have been calculated in-place by the call to `simulation_kernel`, whilst new 
	position data has been written to the buffers `out_x`, `out_y`. We must update the `d_nbodies` data pointers accordingly. */
	swap_float_pointers(&d_nbodies->x, &out_x);
	swap_float_pointers(&d_nbodies->y, &out_y);
}

/* Functions for parsing Command Line Arguments
The expected arguments are: "nbody.exe N D M [-i I] [-f input_file]" */
void print_help() {
	printf("USAGE: \"nbody.exe N D M [-i I] [-f input_file]\", where\n");
	printf("              N  is the number of bodies to simulate.\n");
	printf("              D  is the integer dimension of the activity grid. The Grid has D*D locations.\n");
	printf("              M  is the operation mode, either `CPU` or `OPENMP`\n");
	printf("         [-i I]  [OPTIONAL] Specifies number `I` of simulation iterations to perform. Visualisation mode is used when `-i` flag not set.\n");
	printf("[-f input_file]  [OPTIONAL] Specifies an input file with an initial `N` bodies of data. A random initial state will be generated when `-f` flag not set.\n");
}

void parseNDM(const char* argv[3]) {
	N = parse_str_as_uint(argv[0]);
	checkLastError("Error parsing argument for `N` to `int`");
	if (N == 0) {
		fprintf(stderr, "Error: Argument \"%s\" for number of bodies `N` parsed as 0.\n", argv[0]);
		print_help();
		exit(EXIT_FAILURE);
	}
	D = parse_str_as_uint(argv[1]);
	checkLastError("Error parsing argument for `D` to `int`");
	if (strcmp(argv[2], "CPU") == 0) {
		M = CPU;
	}
	else if (strcmp(argv[2], "OPENMP") == 0) {
		M = OPENMP;
	}
	else if (strcmp(argv[2], "CUDA") == 0) {
		M = CUDA;
	}
	else {
		fprintf(stderr, "Error: Unexpected value %s for operation mode `M` (case sensitive).\n", argv[3]);
		print_help();
		exit(EXIT_FAILURE);
	}
}

void parse_one_option(const char* options[2]) {
	if (strcmp(options[0], "-i") == 0) {
		I = parse_str_as_uint(options[1]);
		checkLastError("Error parsing argument for `I` to `int`");
	}
	else if (strcmp(options[0], "-f") == 0) {
		f_flag = 5;
	}
	else { // Invalid option flag
		fprintf(stderr, "Error: Unexpected optional arguments/flags received.\n");
		print_help();
		exit(EXIT_FAILURE);
	}
}

void parse_two_options(const char* options[4]) {
	if ((strcmp(options[0], "-i") == 0) && (strcmp(options[2], "-f") == 0)) {
		I = parse_str_as_uint(options[1]);
		checkLastError("Error parsing argument for `I` to `int`");
		f_flag = 7;
	}
	else if ((strcmp(options[0], "-f") == 0) && (strcmp(options[2], "-i") == 0)) {
		I = parse_str_as_uint(options[3]);
		checkLastError("Error parsing argument for `I` to `int`");
		f_flag = 5;
	}
	else { // Invalid option flag combination
		fprintf(stderr, "Error: Unexpected combination of optional arguments/flags received.\n");
		print_help();
		exit(EXIT_FAILURE);
	}
}

unsigned int parse_str_as_uint(const char * str) {
	if (isdigit(str[0]) == 0) { // In particular, this excludes leading minus sign/negative input values.
		fprintf(stderr, "Error parsing %s as `int`: First char not decimal digit (negative values not permitted).\n", str);
		print_help();
		exit(EXIT_FAILURE);
	}
	unsigned int val; // To hold parsed `unsigned int` value
	char* pEnd; // Pointer to first character after number in `str`
	val = (unsigned int)strtol(str, &pEnd, 10); // Convert string to long integer in base 10. Set `pEnd`.
	if (pEnd[0] != '\0') { // Check for extra characters in `str` after initial number (can include decimal point)
		fprintf(stderr, "Error: Unexpected characters in string %s when parsing to `int`.\n", str);
		print_help();
		exit(EXIT_FAILURE);
	} 
	return val;
}

/* Functions for reading input files */
void read_nbody_file(const char* filename, const int N) {
	FILE* f; // Input file handle
	char line_buffer[BUFFER_SIZE]; // Buffer to hold lines read from file
	char* ptr_ch = NULL; // Pointer to track character position when reading `line_buffer` string
	int line_number = 0; // Keep track of line number for error messaging
	int body_count = 0; // Count of number of body data lines read to ensure it matches `N`

	f = fopen(filename, "r"); // Open the file in read-only mode
	if (f == NULL) {
		fprintf(stderr, "Error opening file '%s' for reading\n", filename);
		exit(EXIT_FAILURE);
	}

	/* Read file line by line with `fgets` function. See http://www.cplusplus.com/reference/cstdio/fgets/ for reference
	Reads from file into buffer until (soonest of) either `\n` or `EOF` is read, or `BUFFER_SIZE-1` characters read */
	while (fgets(line_buffer, BUFFER_SIZE, f) != NULL) {
		line_number++; // Increment count of lines read
		if (line_buffer[0] == '#') { // If first char in line is `#` skip to next line to ignore comments
			continue;
		}
		if (line_buffer[strlen(line_buffer) - 1] != '\n') { // If last char read from file is not '\n', the line is too long
			// This checks that a full line of data was written from file to buffer when not a comment line
			fprintf(stderr, "Error reading line %u: Line length exceeds buffer size of %d characters\n", line_number, BUFFER_SIZE);
			exit(EXIT_FAILURE);
		}

		/* Read the line of data into `h_nbodies`, using comma character `,` as delimiter to separate data values 
		This could be considered as an unrolled while loop over commas counted using `strchr` calls with nontrivial control flow 
		The use of `ptr_ch` as a separate variable from `line_buffer` could probably be removed. */
		ptr_ch = line_buffer; // Place `ptr_ch` at the start of the line to be read
		/* Use `strchr` to search through the line starting at position `ptr_ch` to find the next comma `,` character
		Returns `NULL` pointer if no comma `,` character found in line after position `ptr_ch`
		See http://www.cplusplus.com/reference/cstring/strchr/ for reference */
		if ((strchr(ptr_ch, ',') == NULL)) { // Check for comma after first data value
			fprintf(stderr, "Error reading line %u: No data delimiters (`,`) detected\n", line_number);
			exit(EXIT_FAILURE);
		}
		else { // This appears to be a valid data line. Don't write past memory bounds for `h_nbodies`!
			if (body_count > N-1) { // Throw an error if we have already read `N` or more data rows
				fprintf(stderr, "Error reading line %u: Num bodies in file exceeds input N (%d)\n", line_number, N);
				exit(EXIT_FAILURE);
			} 
			/* Read `float x` value or randomly generate if data missing */
			// Move `ptr_ch` past any whitespace, then check if the string starts with `[+-]?[0-9]+`
			while (isspace(ptr_ch[0])) {
				ptr_ch++;
			}
			// If string matches `[+-]?[0-9]+.*` after preceding whitespace, parse with `strtod`
			if (isdigit(ptr_ch[0]) || (((ptr_ch[0] == '+') || (ptr_ch[0] == '-')) && isdigit(ptr_ch[1]))) {
				// Parse and store `x` value, then update `ptr_ch` to point to end of number
				h_nbodies->x[body_count] = (float)strtod(ptr_ch, &ptr_ch);
				checkLastError("Error parsing `x` data to `float`");
				// Check there are no further digits before the comma at `strchr(ptr_ch, ',')`
				if ((strpbrk(ptr_ch, "0123456789") < strchr(ptr_ch, ',')) && (strpbrk(ptr_ch, "0123456789") != NULL)) {
					fprintf(stderr, "Error reading line %u: Unexpected format when parsing `x` data to float\n", line_number);
					exit(EXIT_FAILURE);
				}
			}
			else { // Decide data missing or corrupted - means we ignore strings like ".5" and "-.2"
				h_nbodies->x[body_count] = (float)rand() / RAND_MAX; // Random position in [0,1]
			}
			ptr_ch = strchr(ptr_ch, ',') + 1; // Update `ptr_ch` to start after the 1st comma
		}
		if ((strchr(ptr_ch, ',') == NULL)) { // Check for comma after second data value
			fprintf(stderr, "Error reading line %u: Only 1 data delimiter (`,`) detected\n", line_number);
			exit(EXIT_FAILURE);
		}
		else { /* Read `float y` value or randomly generate if missing */
			// Move `ptr_ch` past any whitespace, then check if the string starts with `[+-]?[0-9]+`
			while (isspace(ptr_ch[0])) {
				ptr_ch++;
			}
			// If string matches `[+-]?[0-9]+.*` after preceding whitespace, parse with `strtod`
			if (isdigit(ptr_ch[0]) || (((ptr_ch[0] == '+') || (ptr_ch[0] == '-')) && isdigit(ptr_ch[1]))) {
				// Parse and store `y` value, then update `ptr_ch` to point to end of number
				h_nbodies->y[body_count] = (float)strtod(ptr_ch, &ptr_ch);
				checkLastError("Error parsing `y` data to `float`");
				// Check there are no further digits before the comma at `strchr(ptr_ch, ',')`
				if ((strpbrk(ptr_ch, "0123456789") < strchr(ptr_ch, ',')) && (strpbrk(ptr_ch, "0123456789") != NULL)) {
					fprintf(stderr, "Error reading line %u: Unexpected format when parsing `y` data to float\n", line_number);
					exit(EXIT_FAILURE);
				}
			}
			else { // Decide data missing or corrupted - means we ignore strings like ".5" and "-.2"
				h_nbodies->y[body_count] = (float)rand() / RAND_MAX; // Random position in [0,1]
			}
			ptr_ch = strchr(ptr_ch, ',') + 1; // Update `ptr_ch` to start after 2nd comma
		}
		if ((strchr(ptr_ch, ',') == NULL)) { // Check for comma after third data value
			fprintf(stderr, "Error reading line %u: Only 2 data delimiters (`,`) detected\n", line_number);
			exit(EXIT_FAILURE);
		}
		else { /* Read `float vx` value or set to zero if missing */
			// Move `ptr_ch` past any whitespace, then check if the string starts with `[+-]?[0-9]+`
			while (isspace(ptr_ch[0])) {
				ptr_ch++;
			}
			// If string matches `[+-]?[0-9]+.*` after preceding whitespace, parse with `strtod`
			if (isdigit(ptr_ch[0]) || (((ptr_ch[0] == '+') || (ptr_ch[0] == '-')) && isdigit(ptr_ch[1]))) {
				// Parse and store `vx` value, then update `ptr_ch` to point to end of number
				h_nbodies->vx[body_count] = (float)strtod(ptr_ch, &ptr_ch);
				checkLastError("Error parsing `vx` data to `float`");
				// Check there are no further digits before the comma at `strchr(ptr_ch, ',')`
				if ((strpbrk(ptr_ch, "0123456789") < strchr(ptr_ch, ',')) && (strpbrk(ptr_ch, "0123456789") != NULL)) {
					fprintf(stderr, "Error reading line %u: Unexpected format when parsing `vx` data to float\n", line_number);
					exit(EXIT_FAILURE);
				}
			} // Otherwise decide data is missing or corrupted - means strings like ".5" and "-.2" are ignored
			// In this case we don't change `vx` since velocity array filled with zeroes by default
			ptr_ch = strchr(ptr_ch, ',') + 1; // Update `ptr_ch` to start after 3rd comma
		}
		if ((strchr(ptr_ch, ',') == NULL)) { // Check for comma after fourth data value
			fprintf(stderr, "Error reading line %u: Only 3 data delimiters (`,`) detected\n", line_number);
			exit(EXIT_FAILURE);
		}
		else { /* Read `float vy` value or set to zero if missing */
			// Move `ptr_ch` past any whitespace, then check if the string starts with `[+-]?[0-9]+`
			while (isspace(ptr_ch[0])) {
				ptr_ch++;
			}
			// If string matches `[+-]?[0-9]+.*` after preceding whitespace, parse with `strtod`
			if (isdigit(ptr_ch[0]) || (((ptr_ch[0] == '+') || (ptr_ch[0] == '-')) && isdigit(ptr_ch[1]))) {
				// Parse and store `vx` value, then update `ptr_ch` to point to end of number
				h_nbodies->vy[body_count] = (float)strtod(ptr_ch, &ptr_ch);
				checkLastError("Error parsing `vy` data to `float`");
				// Check there are no further digits before the comma at `strchr(ptr_ch, ',')`
				if ((strpbrk(ptr_ch, "0123456789") < strchr(ptr_ch, ',')) && (strpbrk(ptr_ch, "0123456789") != NULL)) {
					fprintf(stderr, "Error reading line %u: Unexpected format when parsing `vy` data to float\n", line_number);
					exit(EXIT_FAILURE);
				}
			} // Otherwise decide data is missing or corrupted - means strings like ".5" and "-.2" are ignored
			// In this case we don't change `vy` since velocity array filled with zeroes by default
			ptr_ch = strchr(ptr_ch, ',') + 1; // Update `ptr_ch` to start after 4th comma
		}
		if ((strchr(ptr_ch, ',') != NULL)) { // Ensure no more commas after fifth data value
			fprintf(stderr, "Error reading line %u: Too many data columns detected (5 expected)\n", line_number);
			exit(EXIT_FAILURE);
		}
		else { // Else read from after the 4th/last comma (`ptr_ch`) to the end of the line
		/* Read `float m` value or set to 1/N if data missing, corrupted, or zero (no massless bodies) */
			if (strtod(ptr_ch, NULL) == 0) { // If zero returned, then input data was either missing, corrupted, or zero
				fprintf(stderr, "Error reading line %u: Mass data missing, corrupted, or set to zero. Replacing with default value (1/N) to avoid massless bodies\n", line_number);
				// Set mass to 1/N to avoid creating massless objects (and divide-by-zero problems later)
				h_nbodies->m[body_count] = (float)1 / N; // Mass distributed equally among N bodies
			}
			else { // Otherwise non-zero `float` value for mass read successfully, so write to `m`
				// Parse and store `m` value, then update `ptr_ch` to point to end of number
				h_nbodies->m[body_count] = (float)strtod(ptr_ch, &ptr_ch);
				checkLastError("Error parsing mass data to `float`");
				if (strpbrk(ptr_ch, "0123456789") != NULL) { // Check there are no further digits before the end of the line
					fprintf(stderr, "Error reading line %u: Unexpected format when parsing mass data\n", line_number);
					exit(EXIT_FAILURE);
				}
			}
		} // One line of nbody data has been read successfully. Increment the body count.
		body_count++;
		// Read new line if not end of file. Thus data file can be terminated with single empty line.
	}
	if (body_count != N) { // Check fails when fewer than N bodies in file
		fprintf(stderr, "Error: Num bodies in file (%u) does not match input N (%d)\n", body_count, N);
		exit(EXIT_FAILURE);
	}
	fclose(f);
}

void checkLastError(const char* msg) {
	if (errno != 0) {
		perror(msg);
		print_help();
		exit(EXIT_FAILURE);
	}
}

void checkCUDAError(const char* msg) {
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}
